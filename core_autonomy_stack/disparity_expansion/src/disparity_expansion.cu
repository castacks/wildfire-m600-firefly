#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cuda_disparity_expansion.h"

__global__
void fg_bg_u(int table_rows, int u_cols, int v_cols, int disp_rows, int disp_cols,
	     float padding, float SCALE, int lut_max_disparity, float pixel_error, float robot_radius,
	     float bg_multiplier, float baseline, float fx,
	     float* disparity, cell* table_u, cell* table_v, float* disparity_fg, float* disparity_bg);

__global__
void fg_bg_v(int table_rows, int u_cols, int v_cols, int disp_rows, int disp_cols,
	     float padding, float SCALE, int lut_max_disparity, float pixel_error, float robot_radius,
	     float bg_multiplier, float baseline, float fx,
	     float* disparity, float* disparity_background, cell* table_u, cell* table_v, float* disparity_fg, float* disparity_bg);

CudaDisparityExpansion::CudaDisparityExpansion(){
  disparity_allocated = false;
}

void CudaDisparityExpansion::upload_LUT(cell* _table_u, cell* _table_v, int _table_rows, int _u_cols, int _v_cols){
  table_rows = _table_rows;
  u_cols = _u_cols;
  v_cols = _v_cols;
  
  /*for(int r = 0; r < table_rows; r++){
    for(int c = 0; c < u_cols; c++){
      printf("%d-%d ", _table_u[r*u_cols + c].idx1, _table_u[r*u_cols + c].idx2);
    }
    printf("\n");
  }*/
  
  hipMalloc(&table_u, _table_rows*_u_cols*sizeof(cell));
  hipMalloc(&table_v, _table_rows*_v_cols*sizeof(cell));
  hipMemcpy(table_u, _table_u, _table_rows*_u_cols*sizeof(cell), hipMemcpyHostToDevice);
  hipMemcpy(table_v, _table_v, _table_rows*_v_cols*sizeof(cell), hipMemcpyHostToDevice);
}

void CudaDisparityExpansion::upload_disparity(unsigned char* data, int rows, int cols){
  disp_rows = rows;
  disp_cols = cols;
  
  if(!disparity_allocated){
    hipMalloc(&disparity, rows*cols*sizeof(float));
    hipMalloc(&disparity_fg, rows*cols*sizeof(float));
    hipMalloc(&disparity_bg, rows*cols*sizeof(float));
    disparity_allocated = true;
  }
  
  hipMemcpy(disparity, data, rows*cols*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(disparity_fg, data, rows*cols*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(disparity_bg, data, rows*cols*sizeof(float), hipMemcpyHostToDevice);
}

void CudaDisparityExpansion::compute_fg_bg(float padding, float SCALE, int lut_max_disparity,
					   float pixel_error, float robot_radius, float bg_multiplier,
					   float baseline, float fx,
					   unsigned char* data, int rows, int cols){
  // upload the disparity image
  disp_rows = rows;
  disp_cols = cols;
  
  if(!disparity_allocated){
    hipMalloc(&disparity, rows*cols*sizeof(float));
    hipMalloc(&disparity_background, rows*cols*sizeof(float));
    hipMalloc(&disparity_fg, rows*cols*sizeof(float));
    hipMalloc(&disparity_bg, rows*cols*sizeof(float));
    disparity_allocated = true;
  }
  
  hipMemcpy(disparity, data, rows*cols*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(disparity_fg, data, rows*cols*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(disparity_bg, data, rows*cols*sizeof(float), hipMemcpyHostToDevice);
  
  // fill out the fg and bg disparity images, iterating through columns/x direction
  dim3 threads_per_block(16);
  dim3 num_blocks_u(disp_rows/threads_per_block.x + 1);
  fg_bg_u<<<num_blocks_u, threads_per_block>>>(table_rows, u_cols, v_cols, disp_rows, disp_cols,
					       padding, SCALE, lut_max_disparity, pixel_error, robot_radius,
					       bg_multiplier, baseline, fx, disparity, table_u, table_v,
					       disparity_fg, disparity_bg);
  
  // update the disparity and background images
  hipMemcpy(disparity, disparity_fg, rows*cols*sizeof(float), hipMemcpyDeviceToDevice);
  hipMemcpy(disparity_background, disparity_bg, rows*cols*sizeof(float), hipMemcpyDeviceToDevice);
  
  // fill out the fg and bg disparity images, iterating through rows/y direction
  dim3 num_blocks_v(disp_cols/threads_per_block.x + 1);
  fg_bg_v<<<num_blocks_v, threads_per_block>>>(table_rows, u_cols, v_cols, disp_rows, disp_cols,
					       padding, SCALE, lut_max_disparity, pixel_error, robot_radius,
					       bg_multiplier, baseline, fx, disparity, disparity_background, table_u, table_v,
					       disparity_fg, disparity_bg);
  
}

void CudaDisparityExpansion::download_fg_bg(unsigned char* fg, unsigned char* bg){
  hipMemcpy(fg, disparity_fg, disp_rows*disp_cols*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(bg, disparity_bg, disp_rows*disp_cols*sizeof(float), hipMemcpyDeviceToHost);
}

__global__
void fg_bg_u(int table_rows, int u_cols, int v_cols, int disp_rows, int disp_cols,
	     float padding, float SCALE, int lut_max_disparity, float pixel_error, float robot_radius,
	     float bg_multiplier, float baseline, float fx,
	     float* disparity, cell* table_u, cell* table_v, float* disparity_fg, float* disparity_bg){
  int v = blockIdx.x*blockDim.x + threadIdx.x;
  
  // return if the index is out of bounds
  if(v >= disp_rows)
    return;
  
  for(int u = disp_cols - 1; u >= 0; u--){
    float disparity_value = disparity[v*disp_cols + u];
    if(!isnan(double(disparity_value)) &&
       ((int(disparity_value*SCALE)+1) < lut_max_disparity) &&
       ((int(disparity_value*SCALE)+1) > 0)){
      // get bounds
      unsigned int u1 = table_u[u_cols*(int(disparity_value*SCALE)+1) + u].idx1; //maybe add pixel_error
      unsigned int u2 = table_u[u_cols*(int(disparity_value*SCALE)+1) + u].idx2;
      
      // find min and max in bounds
      float u_max = 0.f;
      int max_idx = u-1;//0;
      for(int t = u1; t < u2; t++){
	float value = disparity[v*disp_cols + t];
	if(value > u_max){
	  u_max = value;
	  max_idx = t;
	}
      }
      
      // new values
      float disp_new_fg = u_max;
      float disp_to_depth = baseline * fx/u_max;
      float disp_to_depth_orig = disp_to_depth;
      
      // pad
      if(padding < 0.0){
	float range = bg_multiplier * robot_radius;
	float max_depth = 0.0;
	bool found = true;
	int count = 1;
	while(found){
	  found = false;
	  for(int t = u1; t < u2; t++){
	    float val = baseline*fx / disparity[v*disp_cols + t] - disp_to_depth_orig;
	    if(val-val == 0){ // is finite
	      if(val < count*range && val > max_depth){
		found = true;
		max_depth = val;
	      }
	    }
	  }
	  count++;
	}
	disp_to_depth += max_depth;
      }
      else
	disp_to_depth += padding;
      
      float disp_new_bg = baseline * fx/disp_to_depth;
      
      // set values
      for(int t = u1; t < u2; t++){
	int index = v*disp_cols + t;
	disparity_fg[index] = disp_new_fg;
	disparity_bg[index] = disp_new_bg;
      }
      
      // increment u
//      int u_temp = /*u1 + */max_idx;
//      if(u_temp >= u)
//	u = u1;
//      else
//      u = u_temp + 1;
    }
  }
}


__global__
void fg_bg_v(int table_rows, int u_cols, int v_cols, int disp_rows, int disp_cols,
	     float padding, float SCALE, int lut_max_disparity, float pixel_error, float robot_radius,
	     float bg_multiplier, float baseline, float fx,
	     float* disparity, float* disparity_background, cell* table_u, cell* table_v, float* disparity_fg, float* disparity_bg){
  //int v = blockIdx.x*blockDim.x + threadIdx.x;
  int u = blockIdx.x*blockDim.x + threadIdx.x;
  
  // return if the index is out of bounds
  if(u >= disp_cols)
    return;
  
  for(int v = disp_rows - 1; v >= 0; v--){
    float disparity_value = disparity[v*disp_cols + u] + pixel_error;
    if(!isnan(double(disparity_value)) &&
       ((int(disparity_value*SCALE)+1) < lut_max_disparity) &&
       ((int(disparity_value*SCALE)+1) > 0)){
      // get bounds
      unsigned int v1 = table_v[v_cols*(int(disparity_value*SCALE)+1) + v].idx1;
      unsigned int v2 = table_v[v_cols*(int(disparity_value*SCALE)+1) + v].idx2;
      
      // find min and max in bounds
      float v_max = 0.f;
      int max_idx = v-1;//0;
      for(int t = v1; t < v2; t++){
	float value = disparity[t*disp_cols + u];
	if(value > v_max){
	  v_max = value;
	  max_idx = t;
	}
      }
      
      // new values
      float disp_to_depth = baseline * fx/v_max;
      float disp_new_fg = baseline * fx/(disp_to_depth - robot_radius) + pixel_error;
      v_max = 0.f;
      for(int t = v1; t < v2; t++){
	float value = disparity_background[t*disp_cols + u];
	if(value > v_max)
	  v_max = value;
      }
      disp_to_depth = baseline * fx/v_max;
      float disp_to_depth_orig = disp_to_depth;
      
      // pad
      if(padding < 0.0){
	float range = bg_multiplier * robot_radius;
	float max_depth = 0.0;
	bool found = true;
	int count = 1;
	while(found){
	  found = false;
	  for(int t = v1; t < v2; t++){
	    float val = baseline*fx / disparity[t*disp_cols + u] - disp_to_depth_orig;
	    if(val-val == 0){ // is finite
	      if(val < count*range && val > max_depth){
		found = true;
		max_depth = val;
	      }
	    }
	  }
	  count++;
	}
	disp_to_depth += max_depth;
      }
      else
	disp_to_depth += padding;
      
      float disp_new_bg = baseline * fx/(disp_to_depth + robot_radius) - pixel_error;
      disp_new_bg = disp_new_bg < 0.0 ? 0.0001 : disp_new_bg;
      
      // set values
      for(int t = v1; t < v2; t++){
	int index = t*disp_cols + u;
	disparity_fg[index] = disp_new_fg;//max(disp_new_fg, disparity_fg[index]);
	disparity_bg[index] = disp_new_bg;//max(disp_new_bg, disparity_bg[index]);
      }
      
      // increment v
//      int v_temp = /*v1 + */max_idx;
//      if(v_temp >= v)
//	v = v1;
//      else
//	v = v_temp + 1;
    }
  }
}


